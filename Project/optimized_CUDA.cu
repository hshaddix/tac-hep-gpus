// Part 3: Optimized CUDA 

#include <iostream>
#include <hip/hip_runtime.h>


#define DSIZE 512
#define RADIUS 3
#define TILE_SIZE 16  // Tile size for shared memory

// CUDA error-checking utility
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << " - Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

// Initialize matrix with arbitrary values on host
void initializeMatrix(int* matrix) {
    for (int i = 0; i < DSIZE * DSIZE; ++i) {
        matrix[i] = rand() % 10;
    }
}

// Kernel for stencil operation using shared memory
__global__ void stencilKernel(int* matrix, int* result) {
    __shared__ int tile[TILE_SIZE + 2 * RADIUS][TILE_SIZE + 2 * RADIUS];
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = threadIdx.x + RADIUS;
    int ty = threadIdx.y + RADIUS;

    if (x < DSIZE && y < DSIZE) {
        // Load data into shared memory, including halo regions
        tile[ty][tx] = matrix[y * DSIZE + x];
        if (threadIdx.x < RADIUS) {
            tile[ty][tx - RADIUS] = (x >= RADIUS) ? matrix[y * DSIZE + x - RADIUS] : 0;
            tile[ty][tx + TILE_SIZE] = (x + TILE_SIZE < DSIZE) ? matrix[y * DSIZE + x + TILE_SIZE] : 0;
        }
        if (threadIdx.y < RADIUS) {
            tile[ty - RADIUS][tx] = (y >= RADIUS) ? matrix[(y - RADIUS) * DSIZE + x] : 0;
            tile[ty + TILE_SIZE][tx] = (y + TILE_SIZE < DSIZE) ? matrix[(y + TILE_SIZE) * DSIZE + x] : 0;
        }
        __syncthreads();

        // Compute the stencil if within bounds
        if (x >= RADIUS && x < DSIZE - RADIUS && y >= RADIUS && y < DSIZE - RADIUS) {
            int sum = 0;
            for (int i = -RADIUS; i <= RADIUS; i++) {
                for (int j = -RADIUS; j <= RADIUS; j++) {
                    sum += tile[ty + j][tx + i];
                }
            }
            result[y * DSIZE + x] = sum;
        }
    }
}

// Kernel for matrix multiplication using shared memory
__global__ void matrixMultiplyKernel(int* A, int* B, int* C) {
    __shared__ int tileA[TILE_SIZE][TILE_SIZE];
    __shared__ int tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    int sum = 0;
    for (int i = 0; i < DSIZE / TILE_SIZE; ++i) {
        // Load tiles from A and B into shared memory
        tileA[threadIdx.y][threadIdx.x] = A[row * DSIZE + i * TILE_SIZE + threadIdx.x];
        tileB[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * DSIZE + col];
        __syncthreads();

        // Perform partial matrix multiplication for this tile
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < DSIZE && col < DSIZE) {
        C[row * DSIZE + col] = sum;
    }
}

// Utility function to verify the result (simple sum check)
bool verifyResult(int* matrix) {
    int total = 0;
    for (int i = 0; i < DSIZE * DSIZE; ++i) {
        total += matrix[i];
    }
    std::cout << "Matrix sum: " << total << std::endl;
    return true;  // Placeholder verification
}

int main() {
    // Allocate host memory
    int* h_A = new int[DSIZE * DSIZE];
    int* h_B = new int[DSIZE * DSIZE];
    int* h_C = new int[DSIZE * DSIZE];

    initializeMatrix(h_A);
    initializeMatrix(h_B);

    // Allocate device memory
    int *d_A, *d_B, *d_tempA, *d_tempB, *d_C;
    hipMalloc((void**)&d_A, DSIZE * DSIZE * sizeof(int));
    hipMalloc((void**)&d_B, DSIZE * DSIZE * sizeof(int));
    hipMalloc((void**)&d_tempA, DSIZE * DSIZE * sizeof(int));
    hipMalloc((void**)&d_tempB, DSIZE * DSIZE * sizeof(int));
    hipMalloc((void**)&d_C, DSIZE * DSIZE * sizeof(int));

    // Create CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Copy data to device asynchronously
    hipMemcpyAsync(d_A, h_A, DSIZE * DSIZE * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, h_B, DSIZE * DSIZE * sizeof(int), hipMemcpyHostToDevice, stream2);

    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((DSIZE + TILE_SIZE - 1) / TILE_SIZE, (DSIZE + TILE_SIZE - 1) / TILE_SIZE);

    // Launch stencil kernels on matrices A and B asynchronously
    stencilKernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_tempA);
    stencilKernel<<<blocksPerGrid, threadsPerBlock, 0, stream2>>>(d_B, d_tempB);

    // Synchronize streams before proceeding to matrix multiplication
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Launch matrix multiplication kernel on a single stream
    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_tempA, d_tempB, d_C);

    // Copy result back to host
    hipMemcpy(h_C, d_C, DSIZE * DSIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Verify result
    verifyResult(h_C);

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_tempA);
    hipFree(d_tempB);
    hipFree(d_C);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}

// Output: 
// Matrix sum: 1295711823

// Profiling 

// ==3273902== NVPROF is profiling process 3273902, command: ./optimized_CUDA
// Matrix sum: 1295711823
// ==3273902== Profiling application: ./optimized_CUDA
// ==3273902== Profiling result:
//             Type  Time(%)      Time     Calls       Avg       Min       Max  Name
//  GPU activities:   58.85%  568.15us         1  568.15us  568.15us  568.15us  matrixMultiplyKernel(int*, int*, int*)
//                    17.02%  164.29us         2  82.143us  81.887us  82.399us  [CUDA memcpy HtoD]
//                    14.07%  135.84us         2  67.919us  65.919us  69.919us  stencilKernel(int*, int*)
//                    10.06%  97.118us         1  97.118us  97.118us  97.118us  [CUDA memcpy DtoH]
//       API calls:   95.55%  275.74ms         5  55.147ms  4.4300us  275.35ms  cudaMalloc
//                     2.60%  7.5065ms       228  32.923us     140ns  3.2381ms  cuDeviceGetAttribute
//                     0.73%  2.0952ms         3  698.41us  10.880us  2.0721ms  cudaLaunchKernel
//                     0.69%  1.9904ms         1  1.9904ms  1.9904ms  1.9904ms  cudaMemcpy
//                     0.22%  626.50us         5  125.30us  5.3900us  266.11us  cudaFree
//                     0.12%  342.29us         2  171.14us  149.17us  193.11us  cudaMemcpyAsync
//                     0.04%  112.57us         2  56.286us  54.481us  58.092us  cudaStreamSynchronize
//                     0.02%  55.852us         2  27.926us  4.9710us  50.881us  cudaStreamCreate
//                     0.02%  51.611us         2  25.805us  11.291us  40.320us  cuDeviceGetName
//                     0.01%  25.511us         2  12.755us  7.3500us  18.161us  cudaStreamDestroy
//                     0.01%  22.762us         2  11.381us  4.5210us  18.241us  cuDeviceGetPCIBusId
//                     0.00%  2.8510us         4     712ns     180ns  2.1800us  cuDeviceGet
//                     0.00%  2.4500us         3     816ns     180ns  1.9500us  cuDeviceGetCount
//                     0.00%  1.0210us         2     510ns     410ns     611ns  cuDeviceTotalMem
//                     0.00%     930ns         2     465ns     380ns     550ns  cuDeviceGetUuid
//                     0.00%     500ns         1     500ns     500ns     500ns  cuModuleGetLoadingMode
